// ######################################################
// ##  姓名: 刘羽丰
// ##  文件说明: 第六版，与第五版不同的是这版在输入数组上使用
// ##           二维纹理内存，利用高维空间局部性加快运算，
// ##            log数组使用常量内存存储。
// ######################################################

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h> 
 
#include <cmath>

// 声明常量内存
__constant__ float mylog[25];
// ######################################################
// ##  INIT函数,将主机端的两数组进行初始化
// ##  A_host: 输入的矩阵,初始化时随机0-15
// ##  B_host: 输出的矩阵,初始化时初始为0
// ##  ROWS,COLS分别为矩阵的高和宽
// ######################################################
void INIT(int* A_host,float* B_host,int ROWS,int COLS){
    // srand(time(NULL));
    for(int i = 0; i < ROWS; i++){
        for(int j = 0; j < COLS; j++){
            A_host[i*COLS+j] = rand() % 16;
            B_host[i*COLS+j] = 0;
        }
    }
}

// ######################################################
// ##  核函数,计算二维数组中以每个元素为中心的熵
// ##  A: 输入的矩阵
// ##  B: 输出的矩阵
// ##  rows,cols分别为矩阵的高和宽
// ######################################################
__global__ void cal_entropy(hipTextureObject_t tex,float *B, int rows, int cols){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < rows*cols){
        // 变换得到坐标
        int row = idx / cols;
        int col = idx - row*cols;
        // 避免大量重复if，计算窗口四边界
        int up = max(row-2,0);
        int down = min(row+2,rows-1);
        int left = max(col-2,0);
        int right = min(col+2,cols-1);
        // 利用char存储降低寄存器压力
        char digit[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
        // printf("(%d,%d) l:%d,r:%d,u:%d,d:%d\n",row,col,left,right,up,down);
        // 根据窗口四边界可直接得到窗口大小
        int count = (right-left+1)*(down-up+1);

        // 记录熵
        float result = 0;
        // 遍历周围元素并统计
        for(int i = up; i <= down; i++){
            for(int j = left; j <= right; j++){
                ++digit[tex2D<int>(tex,j,i)];
            }
        }
        // 计算熵
        for(int i = 0; i < 16; i++){
            if(digit[i] != 0){
                result += (((float)digit[i])/count)*(mylog[digit[i]-1]-mylog[count-1]);
            }
        }
        // 读入到输出矩阵
        B[idx] = -result;
        // printf("c:%d idx:%d r:%f\n",count,idx,B[idx]);
    }
}

//主函数
int main(int argc,char *argv[])
{   
    int i;
    // 矩阵高度
    int ROWS = 5;
    // 矩阵宽度
    int COLS = 5;
    // 一个块的线程数
    int block = 256;

    for(i = 1; i < argc; i++)
    {
        if(i == 1){
            ROWS = atoi(argv[i]);
        }
        else if(i == 2){
            COLS = atoi(argv[i]);
        }
        else if(i == 3){
            block = atoi(argv[i]);
        }
    }
    // 输入的二维数组,值为0-15
    int Bytes = ROWS*COLS*sizeof(int);
    // 输出的二维数组,元素类型为float
    int FBytes = ROWS*COLS*sizeof(float);
    //开辟主机内存
    int* A_host = (int*)malloc(Bytes);
    float* B_host = (float*)malloc(FBytes);
    // 初始化
    INIT(A_host,B_host,ROWS,COLS);

    //开辟设备内存
    int* A_dev = NULL;
    float* B_dev = NULL;
    hipMalloc((void**)&A_dev, Bytes);
    hipMalloc((void**)&B_dev, FBytes);
    // size_t pitch,tex_ofs;
    // cudaMallocPitch((void**)&A_dev,&pitch,COLS*sizeof(int),ROWS);
    // cudaMemcpy2D(A_dev,pitch,A_host,COLS*ROWS*sizeof(int),COLS*sizeof(int),ROWS,cudaMemcpyDeviceToHost);
    // tex.normalized = false;
    // cudaBindTexture2D(&tex_ofs, &tex, A_dev, &tex.channelDesc,COLS, ROWS, pitch);
    hipArray* carray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    hipMallocArray(&carray, &channelDesc, COLS, ROWS);
    hipMemcpy2DToArray(carray, 0, 0, A_host, COLS*sizeof(int), COLS*sizeof(int), ROWS, hipMemcpyHostToDevice);

    //输入数据从主机内存拷贝到设备内存
    // cudaMemcpy(A_dev, A_host, Bytes, cudaMemcpyHostToDevice);
    hipMemcpy(B_dev, B_host, FBytes, hipMemcpyHostToDevice);
    // log数组
    const float my_log[25] = {
        0.000000,
        0.693147,
        1.098612,
        1.386294,
        1.609438,
        1.791759,
        1.945910,
        2.079442,
        2.197225,
        2.302585,
        2.397895,
        2.564949,
        2.639057,
        2.708050,
        2.772589,
        2.833213,
        2.890372,
        2.944439,
        2.995732,
        3.044522,
        3.091042,
        3.135494,
        3.178054};
    // 拷贝log数组到常量内存中
    hipMemcpyToSymbol(HIP_SYMBOL(mylog), (const float*)my_log, sizeof(my_log));
    // 纹理内存
    // 描述通道信息,绑定纹理到array上
    hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = carray;
    // 设置纹理为只读
    hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    // 创建纹理对象
    hipTextureObject_t tex;
    hipCreateTextureObject(&tex,&resDesc,&texDesc,NULL);

    //GPU计时
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //运行程序
    cal_entropy<<<(COLS*ROWS-1)/block+1,block>>>(tex,B_dev, ROWS, COLS);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //输出GPU执行时间
    printf("gpu_time:%fms\n",elapsedTime);
    // 销毁纹理对象
    hipDestroyTextureObject(tex);
    //释放内存
    hipFree(B_dev);
    hipFree(A_dev);
    free(B_host);
    free(A_host);
    return 0;
}